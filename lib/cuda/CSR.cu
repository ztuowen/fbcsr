//
// Created by joe on 4/5/16.
//

#include"../CSR.h"
#include"cuHelper.h"
#include<hipsparse.h>

extern "C" void csr_memCpy(csr *src, csr *dst, enum DeviceCopyDIR dir) {
    dst->m = src->m;
    dst->n = src->n;
    dst->nnz = src->nnz;
    memCopy((void **) &(dst->ptr), (void *) src->ptr, sizeof(int) * (dst->n + 1), dir);
    memCopy((void **) &(dst->indx), (void *) src->indx, sizeof(int) * (src->ptr[src->n]), dir);
    memCopy((void **) &(dst->val), (void *) src->val, sizeof(elem_t) * (src->ptr[src->n]), dir);
}

extern "C" void csr_CUDA_SpMV(csr *m, vector *v, vector *r) {
    hipsparseMatDescr_t descr = 0;
    cuSparseCheck(hipsparseCreateMatDescr(&descr));
    cuSparseCheck(hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL));
    cuSparseCheck(hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO));
    hipsparseHandle_t handle;
    cuSparseCheck(hipsparseCreate(&handle));
    elem_t unit = 1;
    cuSparseCheck(
            hipsparseScsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, m->n, m->m, m->nnz, &unit, descr, m->val, m->ptr,
                           m->indx, v->val, &unit, r->val));
    cuSparseCheck(hipsparseDestroy(handle));
    cuSparseCheck(hipsparseDestroyMatDescr(descr));
}

extern "C" void csr_CUDA_destroy(void *c) {
    csr *cc = (csr *) c;
    safeCudaFree(cc->val);
    safeCudaFree(cc->indx);
    safeCudaFree(cc->ptr);
}
