#include "hip/hip_runtime.h"
//
// Created by joe on 4/4/16.
//

#include"../UBCSR.h"
#include"cuHelper.h"

void ubcsrSingle_memCpy(ubcsr *src, ubcsr *dst, enum DeviceCopyDIR dir) {
    dst->n = src->n;
    dst->m = src->m;
    assert(dst->c == src->c);
    assert(dst->r == src->r);
    dst->nnz = src->nnz;
    dst->nr = src->nr;
    dst->nb = src->nb;
    memCopy((void **) &(dst->rptr), (void *) src->rptr, sizeof(int) * src->nr, dir);
    memCopy((void **) &(dst->bptr), (void *) src->bptr, sizeof(int) * (src->nr + 1), dir);
    memCopy((void **) &(dst->val), (void *) src->val, sizeof(elem_t) * src->nb * src->c * src->r, dir);
    memCopy((void **) &(dst->bindx), (void *) src->bindx, sizeof(int) * (src->nb), dir);
}

extern "C" void ubcsr_memCpy(list *src, list *dst, enum DeviceCopyDIR dir) {
    while (src != NULL && dst != NULL) {
        ubcsrSingle_memCpy((ubcsr *) list_get(src), (ubcsr *) list_get(dst), dir);
        src = list_next(src);
        dst = list_next(dst);
    }
    assert(dst == src);
}

/*  CPU Reference
 *   int i, j, k, l;
 *   int indx = 0;
 *   assert(u->m == v->n);
 *   assert(u->n == r->n);
 *   for (i = 0; i < u->nr; ++i)
 *       for (j = u->bptr[i]; j < u->bptr[i + 1]; ++j) {
 *           for (k = 0; k < u->r; ++k)
 *               for (l = 0; l < u->c; ++l, ++indx)
 *                   r->val[k + u->rptr[i]] += v->val[l + u->bindx[j]] * u->val[indx];
 *       }
 */
__global__ void ubcsrSingle_CUDA_SpMV_krnl(ubcsr u, vector v, vector r) {

}

void ubcsrSingle_CUDA_SpMV(ubcsr *u, vector *v, vector *r) {
    dim3 grid(1, 1), block(1, 1);
    ubcsrSingle_CUDA_SpMV_krnl << < grid, block >> > (*u, *v, *r);
}

extern "C" void ubcsr_CUDA_SpMV(list *l, vector *v, vector *r) {
    ubcsr *u;
    while (l != NULL) {
        u = (ubcsr *) list_get(l);
        if (u->optKernel == NULL)
            ubcsrSingle_CUDA_SpMV(u, v, r);
        else {
            ubcsrSingle_SpMVKernel krnl = (ubcsrSingle_SpMVKernel) u->optKernel;
            krnl(u, v, r);
        }
        l = list_next(l);
    }
}

extern "C" void ubcsr_CUDA_destroy(void *u) {
    ubcsr *uu = (ubcsr *) u;
    safeCudaFree(uu->rptr);
    safeCudaFree(uu->val);
    safeCudaFree(uu->bindx);
    safeCudaFree(uu->bptr);
}
