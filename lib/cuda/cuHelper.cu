#include"cuHelper.h"

void memCopy(void **dst, void *src, size_t numBytes, enum DeviceCopyDIR dir) {
    switch (dir) {
        case cpyHostToDevice: cuCheck(hipMalloc(dst, numBytes));
            cuCheck(hipMemcpy(*dst, src, numBytes, hipMemcpyHostToDevice));
            break;
        case cpyDeviceToHost:
            *dst = malloc(numBytes);
            cuCheck(hipMemcpy(*dst, src, numBytes, hipMemcpyDeviceToHost));
            break;
        default:
            fprintf(stderr, "Unexpected memcpy");
            exit(-1);
    }
}
