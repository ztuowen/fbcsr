#include"cuHelper.h"

void memCopy(void *src,void **dst,int numBytes,enum DeviceCopyDIR dir){
    switch (dir){
        case cpyHostToDevice:
            cuCheck(hipMalloc(dst,numBytes));
            cuCheck(hipMemcpy(*dst,src,numBytes,hipMemcpyHostToDevice));
            break;
        case cpyDeviceToHost:
            *dst = malloc(numBytes);
            cuCheck(hipMemcpy(*dst,src,numBytes,hipMemcpyDeviceToHost));
            break;
        default:
            fprintf(stderr,"Unexpected memcpy");
            exit(-1);
    }
}
