#include"cuHelper.h"

void memCopy(void **dst, void *src, int numBytes, enum DeviceCopyDIR dir) {
    switch (dir) {
        case cpyHostToDevice: cuCheck(hipMalloc(dst, numBytes));
            cuCheck(hipMemcpy(*dst, src, numBytes, hipMemcpyHostToDevice));
            break;
        case cpyDeviceToHost:
            *dst = malloc(numBytes);
            cuCheck(hipMemcpy(*dst, src, numBytes, hipMemcpyDeviceToHost));
            break;
        default:
            fprintf(stderr, "Unexpected memcpy");
            exit(-1);
    }
}
