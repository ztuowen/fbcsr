//
// Created by joe on 4/6/16.
//

#include"prefix.h"
#include"testlib/vector_gen.h"
#include"CSR.h"
#include"FBCSR.h"
#include"FBCSR_krnl.h"

#define TOTALRUNS 1000

typedef void (*testFunc)(void);

int main(int argc, char **argv) {
    if (argc < 2) {
        fprintf(stderr, "USAGE: %s <matrix.csr> <opt>", argv[0]);
        return -1;
    }
    int opt = 1;
    if (argc > 2)
        switch (argv[2][0]) {
            case 'd':
                opt = 0;
                break;
            case 'g':
                opt = 2;
                break;
            default:
                opt = 1;
        }
    csr c;
    vector vec;
    vector ref;

    csr_readFile(argv[1], &c);
    vector_gen_random(&vec, c.m, NULL);
    vector_init(&ref, c.n);

    // Make reference
    {
        vector cuv;
        vector cur;
        csr cum;
        vector_init(&ref, c.n);
        csr_memCpy(&c, &cum, cpyHostToDevice);

        vector_memCpy(&vec, &cuv, cpyHostToDevice);
        vector_memCpy(&ref, &cur, cpyHostToDevice);

        csr_CUDA_SpMV(&cum, &cuv, &cur);

        vector_destroy(&ref);
        vector_memCpy(&cur, &ref, cpyDeviceToHost);

        csr_CUDA_destroy(&cum);
        vector_CUDA_destroy(&cuv);
        vector_CUDA_destroy(&cur);
    }
    // FBCSR
    {
        list *l = NULL;
        list *cul = NULL;
        vector cuv;
        vector cur;
        fbcsr *f;
        csr *rem, curem;
        vector res;
        float eltime;
        hipEvent_t st, ed;
        hipEventCreate(&st);
        hipEventCreate(&ed);
        vector_init(&res, c.n);
        f = (fbcsr *) malloc(sizeof(fbcsr));
        fbcsr_makeEmpty(f, c.n, c.m, 32, 1, 32, 0.4, NULL, (void *) fbcsr_row);
        l = list_add(l, f);
        f = (fbcsr *) malloc(sizeof(fbcsr));
        fbcsr_makeEmpty(f, c.n, c.m, 1, 32, 32, 0.4, NULL, (void *) fbcsr_backwardSlash);
        l = list_add(l, f);
        f = (fbcsr *) malloc(sizeof(fbcsr));
        fbcsr_makeEmpty(f, c.n, c.m, 1, 32, 32, 0.4, NULL, (void *) fbcsr_forwardSlash);
        l = list_add(l, f);
        f = (fbcsr *) malloc(sizeof(fbcsr));
        fbcsr_makeEmpty(f, c.n, c.m, 1, 32, 32, 0.3, NULL, (void *) fbcsr_column);
        l = list_add(l, f);

        f = (fbcsr *) malloc(sizeof(fbcsr));
        fbcsr_makeEmpty(f, c.n, c.m, 32, 1, 32, 0.4, (void *) fbcsr_row_krnl_32, (void *) fbcsr_row);
        cul = list_add(cul, f);
        f = (fbcsr *) malloc(sizeof(fbcsr));
        fbcsr_makeEmpty(f, c.n, c.m, 1, 32, 32, 0.4, (void *) fbcsr_bslash_krnl_32, (void *) fbcsr_backwardSlash);
        cul = list_add(cul, f);
        f = (fbcsr *) malloc(sizeof(fbcsr));
        fbcsr_makeEmpty(f, c.n, c.m, 1, 32, 32, 0.4, (void *) fbcsr_fslash_krnl_32, (void *) fbcsr_forwardSlash);
        cul = list_add(cul, f);
        f = (fbcsr *) malloc(sizeof(fbcsr));
        fbcsr_makeEmpty(f, c.n, c.m, 1, 32, 32, 0.3, (void *) fbcsr_col_krnl_32, (void *) fbcsr_column);
        cul = list_add(cul, f);

        rem = csr_fbcsr(&c, l);

        vector_memCpy(&vec, &cuv, cpyHostToDevice);
        vector_memCpy(&res, &cur, cpyHostToDevice);
        csr_memCpy(rem, &curem, cpyHostToDevice);
        fbcsr_memCpy(l, cul, cpyHostToDevice);

        if (opt) {
            hipEventRecord(st, 0);
            for (int i = 0; i < TOTALRUNS; ++i) {
                csr_CUDA_SpMV(&curem, &cuv, &cur);
                fbcsr_CUDA_SpMV(cul, &cuv, &cur);
            }
            hipEventRecord(ed, 0);
            hipEventSynchronize(ed);
            hipEventElapsedTime(&eltime, st, ed);

            if (opt == 1)
                printf("%f\n", eltime / TOTALRUNS);
            else
                printf("%f\n", 2 * c.nnz / (eltime * (1000000 / TOTALRUNS)));
        } else {
            float cnt = 0;
            list *ll = l;
            while (ll != NULL) {
                fbcsr *f = (fbcsr *) list_get(ll);
                cnt += f->nnz;
                printf("%d\t", f->nnz);
                ll = list_next(ll);
            }
            printf("%f\t%f\n", cnt, cnt / c.nnz * 100);
        }
        vector_memCpy(&res, &cur, cpyHostToDevice);

        csr_CUDA_SpMV(&curem, &cuv, &cur);
        fbcsr_CUDA_SpMV(cul, &cuv, &cur);

        vector_destroy(&res);
        vector_memCpy(&cur, &res, cpyDeviceToHost);

        if (!vector_equal(&ref, &res)) {
            fprintf(stderr, "Result mismatch\n");
        }

        hipEventDestroy(st);
        hipEventDestroy(ed);
        list_destroy(l, fbcsr_destroy);
        csr_destroy(rem);
        free(rem);
        vector_destroy(&res);
        csr_CUDA_destroy(&curem);
        vector_CUDA_destroy(&cuv);
        vector_CUDA_destroy(&cur);
        list_destroy(cul, fbcsr_CUDA_destroy);
    }
    csr_destroy(&c);
    vector_destroy(&vec);
    vector_destroy(&ref);
    return 0;
}

