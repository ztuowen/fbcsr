//
// Created by joe on 4/6/16.
//

#include"prefix.h"
#include"testlib/vector_gen.h"
#include"CSR.h"
#include"testlib/fix_csr.h"
#include"FBCSR.h"
#include"FBCSR_krnl.h"

#define TOTALRUNS 1000

typedef void (*testFunc)(void);

int main(int argc, char **argv) {
    if (argc < 2) {
        fprintf(stderr, "USAGE: %s <matrix.csr> <opt>", argv[0]);
        return -1;
    }
    int opt = 1;
    if (argc > 2)
        switch (argv[2][0]) {
            case 'd':
                opt = 0;
                break;
            case 'g':
                opt = 2;
                break;
            case 'b':
                opt = 3;
                break;
            default:
                opt = 1;
        }
    csr c;
    vector vec;
    vector ref;

    csr_readFile(argv[1], &c);
    fix_csr(&c);
    vector_gen(&vec, c.m, NULL);
    vector_init(&ref, c.n);

    // Make reference
    {
        vector cuv;
        vector cur;
        csr cum;
        vector_init(&ref, c.n);
        csr_memCpy(&c, &cum, cpyHostToDevice);

        vector_memCpy(&vec, &cuv, cpyHostToDevice);
        vector_memCpy(&ref, &cur, cpyHostToDevice);

        csr_CUDA_SpMV(&cum, &cuv, &cur);

        vector_destroy(&ref);
        vector_memCpy(&cur, &ref, cpyDeviceToHost);

        csr_CUDA_destroy(&cum);
        vector_CUDA_destroy(&cuv);
        vector_CUDA_destroy(&cur);
    }
    // FBCSR
    {
        list *l = NULL;
        list *cul = NULL;
        vector cuv;
        vector cur;
        fbcsr *f;
        csr *rem, curem;
        vector res;
        float eltime;
        hipEvent_t st, ed;
        hipEventCreate(&st);
        hipEventCreate(&ed);
        vector_init(&res, c.n);
        f = (fbcsr *) malloc(sizeof(fbcsr));
        fbcsr_makeEmpty(f, c.n, c.m, 32, 1, 32, 0.6, NULL, (void *) fbcsr_row);
        l = list_add(l, f);
        f = (fbcsr *) malloc(sizeof(fbcsr));
        fbcsr_makeEmpty(f, c.n, c.m, 1, 32, 32, 0.6, NULL, (void *) fbcsr_backwardSlash);
        l = list_add(l, f);
        f = (fbcsr *) malloc(sizeof(fbcsr));
        fbcsr_makeEmpty(f, c.n, c.m, 1, 32, 32, 0.6, NULL, (void *) fbcsr_forwardSlash);
        l = list_add(l, f);
        f = (fbcsr *) malloc(sizeof(fbcsr));
        fbcsr_makeEmpty(f, c.n, c.m, 1, 32, 32, 0.6, NULL, (void *) fbcsr_column);
        l = list_add(l, f);
        f = (fbcsr *) malloc(sizeof(fbcsr));
        fbcsr_makeEmpty(f, c.n, c.m, 32, 32, 1024, 0.4, NULL, (void *) fbcsr_square);
        l = list_add(l, f);

        f = (fbcsr *) malloc(sizeof(fbcsr));
        fbcsr_makeEmpty(f, c.n, c.m, 32, 1, 32, 0.6, (void *) fbcsr_row_krnl_32, (void *) fbcsr_row);
        cul = list_add(cul, f);
        f = (fbcsr *) malloc(sizeof(fbcsr));
        fbcsr_makeEmpty(f, c.n, c.m, 1, 32, 32, 0.6, (void *) fbcsr_bslash_krnl_32, (void *) fbcsr_backwardSlash);
        cul = list_add(cul, f);
        f = (fbcsr *) malloc(sizeof(fbcsr));
        fbcsr_makeEmpty(f, c.n, c.m, 1, 32, 32, 0.6, (void *) fbcsr_fslash_krnl_32, (void *) fbcsr_forwardSlash);
        cul = list_add(cul, f);
        f = (fbcsr *) malloc(sizeof(fbcsr));
        fbcsr_makeEmpty(f, c.n, c.m, 1, 32, 32, 0.6, (void *) fbcsr_col_krnl_32, (void *) fbcsr_column);
        cul = list_add(cul, f);
        f = (fbcsr *) malloc(sizeof(fbcsr));
        fbcsr_makeEmpty(f, c.n, c.m, 32, 32, 1024, 0.4, (void *) fbcsr_square_krnl, (void *) fbcsr_square);
        cul = list_add(cul, f);

        rem = csr_fbcsr(&c, l);

        vector_memCpy(&vec, &cuv, cpyHostToDevice);
        vector_memCpy(&res, &cur, cpyHostToDevice);
        csr_memCpy(rem, &curem, cpyHostToDevice);
        fbcsr_memCpy(l, cul, cpyHostToDevice);

        if (opt) {
            hipEventRecord(st, 0);
            for (int i = 0; i < TOTALRUNS; ++i) {
                csr_CUDA_SpMV(&curem, &cuv, &cur);
                fbcsr_CUDA_SpMV(cul, &cuv, &cur);
            }
            hipEventRecord(ed, 0);
            hipEventSynchronize(ed);
            hipEventElapsedTime(&eltime, st, ed);
            float cnt = 0;
            list *ll = l;
            while (ll != NULL) {
                fbcsr *f = (fbcsr *) list_get(ll);
                cnt += f->nr * sizeof(int) * 3; // rptr bptr
                cnt += f->nb * sizeof(int); // bindx
                if (f->optKernel == (void *) fbcsr_square_krnl)
                    cnt += f->nb * f->nelem * sizeof(elem_t) + f->nb * 32 * sizeof(elem_t); // val vec
                else if (f->optKernel == (void *) fbcsr_col_krnl_32)
                    cnt += f->nb * (f->nelem + 1) * sizeof(elem_t); // val vec
                else
                    cnt += f->nb * f->nelem * 2 * sizeof(elem_t); // val vec
                cnt += f->nb * f->r * sizeof(elem_t) * 2; // y[i]+=
                ll = list_next(ll);
            }
            cnt += 2 * sizeof(int) * rem->n;     // row pointer
            cnt += 1 * sizeof(int) * rem->nnz; // column index
            cnt += 2 * sizeof(elem_t) * rem->nnz; // A[i,j] and x[j]
            cnt += 2 * sizeof(elem_t) * rem->n;
            switch (opt) {
                case 1:
                default:
                    printf("%f\n", eltime / TOTALRUNS);
                    break;
                case 2:
                    printf("%f\n", 2 * c.nnz / (eltime * (1000000 / TOTALRUNS)));
                    break;
                case 3:
                    printf("%f\n", cnt / (eltime * (1000000 / TOTALRUNS)));
                    break;
            }
        } else {
            float cnt = 0;
            list *ll = l;
            while (ll != NULL) {
                fbcsr *f = (fbcsr *) list_get(ll);
                cnt += f->nnz;
                printf("%d\t", f->nnz);
                ll = list_next(ll);
            }
            printf("%f\t%f\n", cnt, cnt / c.nnz * 100);
        }
        vector_memCpy(&res, &cur, cpyHostToDevice);

        csr_CUDA_SpMV(&curem, &cuv, &cur);
        fbcsr_CUDA_SpMV(cul, &cuv, &cur);

        vector_destroy(&res);
        vector_memCpy(&cur, &res, cpyDeviceToHost);

        if (!vector_equal(&ref, &res)) {
            fprintf(stderr, "Result mismatch\n");
        }

        hipEventDestroy(st);
        hipEventDestroy(ed);
        list_destroy(l, fbcsr_destroy);
        csr_destroy(rem);
        free(rem);
        vector_destroy(&res);
        csr_CUDA_destroy(&curem);
        vector_CUDA_destroy(&cuv);
        vector_CUDA_destroy(&cur);
        list_destroy(cul, fbcsr_CUDA_destroy);
    }
    csr_destroy(&c);
    vector_destroy(&vec);
    vector_destroy(&ref);
    return 0;
}

