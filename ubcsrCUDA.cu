//
// Created by joe on 4/6/16.
//

#include"prefix.h"
#include"testlib/vector_gen.h"
#include"CSR.h"
#include"UBCSR.h"


typedef void (*testFunc)(void);

int main(int argc, char **argv) {
    if (argc < 2) {
        fprintf(stderr, "USAGE: %s <matrix.csr>", argv[0]);
        return -1;
    }
    csr c;
    vector vec;
    vector ref;

    csr_readFile(argv[1], &c);
    vector_gen_random(&vec, c.m, NULL);
    vector_init(&ref, c.n);

    // Make reference
    {
        vector cuv;
        vector cur;
        csr cum;
        vector_init(&ref, c.n);
        csr_memCpy(&c, &cum, cpyHostToDevice);

        vector_memCpy(&vec, &cuv, cpyHostToDevice);
        vector_memCpy(&ref, &cur, cpyHostToDevice);

        csr_CUDA_SpMV(&cum, &cuv, &cur);

        vector_destroy(&ref);
        vector_memCpy(&cur, &ref, cpyDeviceToHost);

        csr_CUDA_destroy(&cum);
        vector_CUDA_destroy(&cuv);
        vector_CUDA_destroy(&cur);
    }
    // UBCSR
    {
        list *l = NULL;
        list *cul = NULL;
        vector cuv;
        vector cur;
        ubcsr *u;
        csr *rem, curem;
        vector res;
        float eltime;
        hipEvent_t st, ed;
        hipEventCreate(&st);
        hipEventCreate(&ed);
        vector_init(&res, c.n);
        u = (ubcsr *) malloc(sizeof(ubcsr));
        ubcsr_makeEmpty(u, c.n, c.m, 1, 2, NULL);
        l = list_add(l, u);

        u = (ubcsr *) malloc(sizeof(ubcsr));
        ubcsr_makeEmpty(u, c.n, c.m, 1, 2, NULL);
        cul = list_add(cul, u);

        rem = csr_ubcsr(&c, l, 0.8);

        vector_memCpy(&vec, &cuv, cpyHostToDevice);
        vector_memCpy(&res, &cur, cpyHostToDevice);
        csr_memCpy(rem, &curem, cpyHostToDevice);
        ubcsr_memCpy(l, cul, cpyHostToDevice);

        hipEventRecord(st, 0);

        csr_CUDA_SpMV(&curem, &cuv, &cur);
        ubcsr_CUDA_SpMV(cul, &cuv, &cur);

        hipEventRecord(ed, 0);
        hipEventSynchronize(ed);
        hipEventElapsedTime(&eltime, st, ed);
        printf("%f\n", eltime);

        vector_destroy(&res);
        vector_memCpy(&cur, &res, cpyDeviceToHost);

        if (!vector_equal(&ref, &res))
            return -1;

        hipEventDestroy(st);
        hipEventDestroy(ed);
        list_destroy(l, ubcsr_destroy);
        csr_destroy(rem);
        free(rem);
        vector_destroy(&res);
        csr_CUDA_destroy(&curem);
        vector_CUDA_destroy(&cuv);
        vector_CUDA_destroy(&cur);
        list_destroy(cul, ubcsr_CUDA_destroy);
    }
    csr_destroy(&c);
    vector_destroy(&vec);
    vector_destroy(&ref);
    return 0;
}

